#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void Inversao_CUDA(unsigned char* Imagem, int canais){
    int x = blockIdx.x;
    int y = blockIdx.y;
    int idx = (x + y * gridDim.x) * canais;

    float r = Imagem[idx + 0]
    float g = Imagem[idx + 1]
    float b = Imagem[idx + 2]

    for(int i = 0; i < canais; i++){
        Imagem[idx + i] = b * 0.299f + g * 0.587f + r * 0.144f;
    }
}

void Imagem_Inversao_CUDA(unsigned char* Input_Imagem, int altura, int largura, int canais){
    unsigned char* Dev_Input_Imagem = NULL;
    //reservar espaco de memoria da gpu
    hipMalloc((void**)&Dev_Input_Imagem, altura * largura * canais);
    //copiar os dados da cpu para gpu
    hipMemcpy(Dev_Input_Imagem, Input_Imagem, altura * largura * canais, hipMemcpyHostToDevice);
    dim3 Grid_Image(largura, altura);
    Inversao_CUDA << <Grid_Image, 2 >> > (Dev_Input_Image, canais);
    //copiar os dados do processamento para cpu que estao na gpu
    hipMemcpy(Input_Imagem, Dev_Input_Image, altura * largura * canais, hipMemcpyDeviceToHost);
    //gpu livre
    hipFree(Dev_Input_Image);
}